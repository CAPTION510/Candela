#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <complex>
#include <iostream>

__global__ void dtrimultipahb(int m, int n, int k, hipDoubleComplex *a, int lda, hipDoubleComplex *b, int ldb, hipDoubleComplex *c, int bias)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= m || j >= n || i < j + bias)
    {
        return;
    }

    hipDoubleComplex temp = make_hipDoubleComplex(0.0, 0.0);

    for (int l = 0; l < k; l++)
    {
        temp = hipCadd(temp, hipCmul(hipConj(a[i * lda + l]), b[j * ldb + l]));
    }

    int ij = j * (m - bias - 1) - (j - 1) * j / 2 + i - bias;
    c[ij] = temp;
}

void gpu_dtrimultipahb(int m, int n, int k, std::complex<double> *a, int lda, std::complex<double> *b, int ldb, std::complex<double> *c, int bias)
{

    hipDoubleComplex *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeof(hipDoubleComplex) * lda * m);
    hipMalloc(&d_b, sizeof(hipDoubleComplex) * ldb * n);
    int dimc = m - bias - n + 1 > 0 ? (m - bias) * n - n * (n - 1) / 2 : (m - bias + 1) * (m - bias) / 2;
    hipMalloc(&d_c, sizeof(hipDoubleComplex) * dimc);

    hipMemcpy(d_a, a, sizeof(hipDoubleComplex) * lda * m, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(hipDoubleComplex) * ldb * n, hipMemcpyHostToDevice);

    dim3 block_size(16, 16);
    dim3 num_blocks((m + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);

    dtrimultipahb<<<num_blocks, block_size>>>(m, n, k, d_a, lda, d_b, ldb, d_c, bias);

    hipMemcpy(c, d_c, sizeof(hipDoubleComplex) * dimc, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
